#include "hip/hip_runtime.h"
#include "vectorAdd.cuh"

__global__ void vectorAdd(float* a, float* b, float* c, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        c[i] = a[i] + b[i];
    }
}

void printResult() {
    constexpr int n = 100000;
    float* a = new float[n];
    float* b = new float[n];
    float* c = new float[n];

    for (int i = 0; i < n; ++i)
    {
        a[i] = i;
        b[i] = i * 2;
    }

    float* d_a;
    float* d_b;
    float* d_c;

    hipMalloc(&d_a, n * sizeof(float));
    hipMalloc(&d_b, n * sizeof(float));
    hipMalloc(&d_c, n * sizeof(float));

    hipMemcpy(d_a, a, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * sizeof(float), hipMemcpyHostToDevice);

    constexpr int blockSize = 256;
    const int numBlocks = (n + blockSize - 1) / blockSize;

    vectorAdd<<<numBlocks, blockSize>>>(d_a, d_b, d_c, n);
    hipMemcpy(c, d_c, n * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < n; ++i)
    {
        std::cout << c[i] << " ";
    }
    std::cout << std::endl;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    delete[] a;
    delete[] b;
    delete[] c;
}